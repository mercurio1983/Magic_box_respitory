#include "hip/hip_runtime.h"
#include "image.h"
#include "gpu_bayer.h"
#include <iostream>
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// raw to rgb increases the width by 3 
#define RGB_width 3 

//using namespace cv;

__global__  void bayerRG(const unsigned char *in, unsigned char* out,int w,int h){
    int x =  ((blockIdx.x*blockDim.x)+ threadIdx.x);
    int y =  ((blockIdx.y*blockDim.y)+ threadIdx.y);

    //unsigned char r,g,b;        
    // 'R'

 		
    if((x %2 == 0) && (y %2 == 0)) {
    	out[y*w*RGB_width+x*RGB_width+0] = (in[y*w + x]);  // r
		out[y*w*RGB_width+x*RGB_width+1] = (in[y*w + x+1]+in[(y+1)*w+x])/2; //g
		out[y*w*RGB_width+x*RGB_width+2] = (in[(y+1)*w +x +1]); //b
    }
    // x is odd and y is even
    else if((x %2 == 1) && (y %2 == 0)){
    
		out[y*w*RGB_width+x*RGB_width+0] = (in[y*w + x+1]); // r 
		out[y*w*RGB_width+x*RGB_width+1] = (in[y*w + x]+in[(y+1)*w+x+1])/2; // g
		out[y*w*RGB_width+x*RGB_width+2] = (in[(y+1)*w +x]); // b
    }
    // x is even and y is odd
    else if((x %2 == 0) && (y %2 == 1)){
    
		out[y*w*RGB_width+x*RGB_width+0] = (in[(y+1)*w + x]); //r 
		out[y*w*RGB_width+x*RGB_width+1] = (in[y*w + x]+in[(y+1)*w+x+1])/2; //g 
		out[y*w*RGB_width+x*RGB_width+2] = (in[y*w +x +1]); //b
    }
    // x is odd and y is odd
    else if((x %2 == 1) && (y %2 == 1)){
    
		out[y*w*RGB_width+x*RGB_width+0] = (in[(y+1)*w + x+1]); //r
		out[y*w*RGB_width+x*RGB_width+1] = (in[y*w + x+1]+in[(y+1)*w+x])/2; //g
		out[y*w*RGB_width+x*RGB_width+2] = (in[y*w +x]); // b
    }
    
    //out[y*w+x+0] = (in[y*w + x]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x])/2; //(in[(y+1)*w +x +1]);
	//out[y*w+x+2] = (in[(y+1)*w +x +1]);
    
    /* block1 part2*/
    //out[y*w+x+5] = (in[y*w + 2+x]); 
	//out[y*w+x+4] = (in[y*w + 2+x+1]+in[(y+1)*w+2+x])/2;
    //out[y*w+x+3] = (in[(y+1)*w +2+x +1]);
    
    /* block1 part3*/
    //out[(y+3)*w+x+2] = (in[(y+2)*w + x]); 
	//out[(y+3)*w+x+1] = (in[(y+2)*w + x+1]+in[(y+3)*w+x])/2;
    //out[(y+3)*w+x+0] = (in[(y+3)*w +x +1]);
    
    /* block1 part4*/
    //out[(y+3)*w+x+7] = (in[(y+2)*w +2+ x]); 
	//out[(y+3)*w+x+6] = (in[(y+2)*w +2+ x+1]+in[(y+3)*w+2+x])/2;
    //out[(y+3)*w+x+5] = (in[(y+3)*w +2+x +1]);
    
    /* block1 part5*/
    //out[y*w+x+2] = (in[y*w + x-657]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x]-657)/2;
    //out[y*w+x+0] = (in[(y+1)*w +x +1-657]);
    
    /* block1 part6*/
    //out[y*w+x+2] = (in[y*w + x-657]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x]-657)/2;
    //out[y*w+x+0] = (in[(y+1)*w +x +1-657]);
    
    /* block1 part7*/
    //out[y*w+x+2] = (in[y*w + x-657]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x]-657)/2;
    //out[y*w+x+0] = (in[(y+1)*w +x +1-657]);
    
    /* block1 part8*/
    //out[y*w+x+2] = (in[y*w + x-657]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x]-657)/2;
    //out[y*w+x+0] = (in[(y+1)*w +x +1-657]);
    
    /* block1 part9*/
    //out[y*w+x+2] = (in[y*w + x-657]); 
	//out[y*w+x+1] = (in[y*w + x+1]+in[(y+1)*w+x]-657)/2;
    //out[y*w+x+0] = (in[(y+1)*w +x +1-657]);
    
    
}

gpu_bayer::gpu_bayer() : size(0),width(0),height(0)
{
	d_in_l=0;
	d_out_l=0; //private variables
	input_l = 0;
	output_l = 0;
	d_in_r=0;
	d_out_r=0; //private variables
	input_r = 0;
	output_r = 0;

	}

gpu_bayer::~gpu_bayer()
{
	//std::cout << "running delete function" << std::endl;
	//[] indicates it is an array of memory otherwise use delete only
	delete[] input_l;  
	delete[] output_l;
	delete[] input_r;  
	delete[] output_r;

}

void gpu_bayer::add_one_image(unsigned char* in_1)
{
		input_l = (unsigned char*)malloc(width*height*1*sizeof(unsigned char)); 
		output_l = (unsigned char*)malloc(width*height*3*sizeof(unsigned char));
		memcpy(input_l,in_1,size); //copy image to memory
}

void gpu_bayer::add_two_image(unsigned char* in_l,unsigned char* in_r)
{
		input_l = (unsigned char*)malloc(width*height*1*sizeof(unsigned char)); 
		output_l = (unsigned char*)malloc(width*height*3*sizeof(unsigned char));
		input_r = (unsigned char*)malloc(width*height*1*sizeof(unsigned char)); 
		output_r = (unsigned char*)malloc(width*height*3*sizeof(unsigned char));
		memcpy(input_l,in_l,size); //copy image to memory
		memcpy(input_r,in_r,size); //copy image to memory
}

unsigned char* gpu_bayer::get_one_image()
{
return output_l;
}

unsigned char* gpu_bayer::get_image_left(){

return output_l;
}

unsigned char* gpu_bayer::get_image_right()
{
return output_r;
}

void gpu_bayer::gpu_bayerRG1(int set_GPU){

	/* allocate to GPU */
	hipSetDevice(set_GPU);
	hipMalloc((void **) &d_in_l, size);
	hipMalloc((void **) &d_out_l, 3*size);
	//dim3 DimBlock(1,1); //20172 blocks 16 threads	
	//dim3 DimBlock(1,2);
	//dim3 DimBlock(2,2);  
	dim3 DimBlock(2,4); 
	
	//if(access_gpu1 == 1){
		//hipSetDevice(set_GPU);
		//hipDeviceEnablePeerAccess(0,0); 
	hipSetDevice(set_GPU);
	hipMemcpy(d_in_l, input_l, size, hipMemcpyHostToDevice); //copy from Host to Device memory
		
	//dim3 DimBlock(16,12); //1681 blocks 192 threads
    //dim3 DimBlock(8,12); //3362 blocks 92 threads
	dim3 DimGrid(width/DimBlock.x,height/DimBlock.y); 
	
	hipSetDevice(set_GPU);
	bayerRG<<<DimGrid,DimBlock>>>(d_in_l,d_out_l,width,height);
	
	hipSetDevice(set_GPU); 
	hipMemcpy(output_l, d_out_l, 3*size, hipMemcpyDeviceToHost); //copy from Device to Host memory
		
	hipDeviceSynchronize();
	//}
	//else{
	
	//printf("no access\n");
	//}
	
	hipSetDevice(set_GPU);
	hipFree(d_in_l); 
	hipFree(d_out_l); 
	
	/* free GPU memory */
	

    

}


void gpu_bayer::gpu_bayerRG2(){
	
	
	
	/* allocate to GPU */
	hipSetDevice(1);
	hipMalloc((void **) &d_in_l, size);
	hipMalloc((void **) &d_out_l, 3*size);
	
	hipSetDevice(2);
	hipMalloc((void **) &d_in_r, size);
	hipMalloc((void **) &d_out_r, 3*size);
	
	dim3 DimBlock(1,1); //20172 blocks 16 threads
	
	//if(access_gpu1 == 1){
		//hipSetDevice(1);
		//hipDeviceEnablePeerAccess(0,0); 
	hipSetDevice(1);
	hipMemcpy(d_in_l, input_l, size, hipMemcpyHostToDevice); //copy from Host to Device memory
	
	hipSetDevice(2);	
	hipMemcpy(d_in_r, input_r, size, hipMemcpyHostToDevice); //copy from Host to Device memory
	
	
	//dim3 DimBlock(16,12); //1681 blocks 192 threads
		//dim3 DimBlock(8,12); //3362 blocks 92 threads
	dim3 DimGrid(width/DimBlock.x,height/DimBlock.y); 
	
	hipSetDevice(1);
	bayerRG<<<DimGrid,DimBlock>>>(d_in_l,d_out_l,width,height);
	
	hipSetDevice(2);	
	bayerRG<<<DimGrid,DimBlock>>>(d_in_r,d_out_r,width,height); 
	
	hipSetDevice(1); 
	hipMemcpy(output_l, d_out_l, 3*size, hipMemcpyDeviceToHost); //copy from Device to Host memory
	
	hipSetDevice(2);
	hipMemcpy(output_r, d_out_r, 3*size, hipMemcpyDeviceToHost); //copy from Device to Host memory
	
	hipDeviceSynchronize();
	//}
	//else{
	
	//printf("no access\n");
	//}
	
	hipSetDevice(1);
	hipFree(d_in_l); 
	hipFree(d_out_l); 
	
	hipSetDevice(2);
	hipFree(d_in_r); 
	hipFree(d_out_r);
	
	/* free GPU memory */
	

    

}






